#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void testKernel(float*x, int len)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if(tid < len) {
		float sum = x[tid];
		int iter = 0;

		while(iter++ < len) {
			sum += 1;
		}
		x[tid] = sum;
	}

}

int main(int argc, char **argv)
{
	const int streamsNum = 2;
	int N=1<<10; // 1K

	if (argc == 2) {
		N = atoi(argv[1]);
	}

	if (argc > 2) {
		fprintf(stderr, "Too many arguments! ./stream_sync N .\n");
		exit(1);
	}


	std::cout << "Running " << N << " (floats) as the input data size." << std::endl;
	std::cout << "Launching " << streamsNum << " cuda streams." << std::endl;
	
	// host 
	float *h_a = NULL;
	float *h_b = NULL;
	hipHostMalloc((void**)&h_a, sizeof(float) * N, hipHostMallocDefault);
	hipHostMalloc((void**)&h_b, sizeof(float) * N, hipHostMallocDefault);

	// init 
	for(int i=0; i<N; i++) {
		h_a[i] = 0;
		h_b[i] = 0;
	}

	// device 
	float*d_a = NULL; 
	float*d_b = NULL; 
	hipMalloc((void**)&d_a, sizeof(float) * N);
	hipMalloc((void**)&d_b, sizeof(float) * N);

	// streams
	hipStream_t streams[streamsNum];
	for(int i=0; i<streamsNum; i++) {
		hipStreamCreate(&streams[i]);
	}

	// h2d
	hipMemcpyAsync(d_a, h_a, sizeof(float)*N, hipMemcpyHostToDevice, streams[0]);
	hipMemcpyAsync(d_b, h_b, sizeof(float)*N, hipMemcpyHostToDevice, streams[1]);

	// kernel
	dim3 block = dim3(128,1,1);
	dim3 grid = dim3((N + block.x - 1) / block.x,1,1);

	testKernel <<< grid, block, 0, streams[0] >>> (d_a, N);
	testKernel <<< grid, block, 0, streams[1] >>> (d_b, N);

	// d2h
	hipMemcpyAsync(h_a, d_a, sizeof(float)*N, hipMemcpyDeviceToHost, streams[0]);
	hipMemcpyAsync(h_b, d_b, sizeof(float)*N, hipMemcpyDeviceToHost, streams[1]);

	hipDeviceSynchronize(); // NOTE: this is needed to make sure prev dev opt is done! 

	int error_a = 0; 
	for(int i=0; i<N; i++) {
		if(h_a[i] != N) {
			printf("h_a[%d] = %f\n",i, h_a[i]);
			error_a += 1;
		}	
	}
	if(error_a == 0) {
		printf("Pass test on h_a!\n");
	}

	int error_b = 0; 
	for(int i=0; i<N; i++) {
		if(h_b[i] != N) {
			printf("h_b[%d] = %f\n",i, h_b[i]);
			error_b += 1;
		}	
	}
	if(error_b == 0) {
		printf("Pass test on h_b!\n");
	}


	// free
	for(int i=0; i<streamsNum; i++) {
		hipStreamDestroy(streams[i]);
	}

	hipFree(d_a);
	hipFree(d_b);

	hipHostFree(h_a);
	hipHostFree(h_b);

	return 0;
}
