#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include "math.h"
#include "hipblas.h"

// ref: https://www.olcf.ornl.gov/tutorials/concurrent-kernels-ii-batched-library-calls/

int main(int argc, char **argv)
{
	const int streamsNum = 2;
	int N=1<<10; // 1K x 1K matrix

	if (argc == 2) {
		N = atoi(argv[1]);
	}

	if (argc > 2) {
		fprintf(stderr, "Too many arguments! ./stream_sync N .\n");
		exit(1);
	}


	std::cout << "Running " << N << " (floats) as the input data size." << std::endl;
	std::cout << "Launching " << streamsNum << " cuda streams." << std::endl;
	
	// host 
	float *h_a = NULL;
	float *h_b = NULL;
	float *h_c = NULL;
	hipHostMalloc((void**)&h_a, sizeof(float) * N * N, hipHostMallocDefault); // N x N
	hipHostMalloc((void**)&h_b, sizeof(float) * N * N, hipHostMallocDefault);
	hipHostMalloc((void**)&h_c, sizeof(float) * N * N, hipHostMallocDefault);

	float *h_x = NULL;
	float *h_y = NULL;
	float *h_z = NULL;
	hipHostMalloc((void**)&h_x, sizeof(float) * N * N, hipHostMallocDefault); // N x N
	hipHostMalloc((void**)&h_y, sizeof(float) * N * N, hipHostMallocDefault);
	hipHostMalloc((void**)&h_z, sizeof(float) * N * N, hipHostMallocDefault);

	// init 
	for(int i=0; i<N; i++) {
		for(int j=0; j<N; j++) {
			int lid = i * N + j;
			if(i == j) {
				h_a[lid] = sin(lid);	
				h_b[lid] = sin(lid);	
				h_c[lid] = cos(lid) * cos(lid);	

				h_x[lid] = sin(lid);	
				h_y[lid] = sin(lid);	
				h_z[lid] = cos(lid) * cos(lid);	

			}
			else{
				h_a[lid] = 0.; 
				h_b[lid] = 0.; 
				h_c[lid] = 0.; 

				h_x[lid] = 0.; 
				h_y[lid] = 0.; 
				h_z[lid] = 0.; 
			}
		}
	}

	// device 
	float*d_a = NULL; 
	float*d_b = NULL; 
	float*d_c = NULL; 
	hipHostMalloc((void**)&d_a, sizeof(float) * N * N, hipHostMallocDefault);
	hipHostMalloc((void**)&d_b, sizeof(float) * N * N, hipHostMallocDefault);
	hipHostMalloc((void**)&d_c, sizeof(float) * N * N, hipHostMallocDefault);

	float*d_x = NULL; 
	float*d_y = NULL; 
	float*d_z = NULL; 
	hipHostMalloc((void**)&d_x, sizeof(float) * N * N, hipHostMallocDefault);
	hipHostMalloc((void**)&d_y, sizeof(float) * N * N, hipHostMallocDefault);
	hipHostMalloc((void**)&d_z, sizeof(float) * N * N, hipHostMallocDefault);

	// streams
	hipStream_t streams[streamsNum];
	for(int i=0; i<streamsNum; i++) {
		hipStreamCreate(&streams[i]);
	}

	float alpha = 1.;
	float beta  = 1.;

    // cublas 
    hipblasHandle_t handle0;
    hipblasCreate(&handle0);

    hipblasHandle_t handle1;
    hipblasCreate(&handle1);

	// set matrices on device
	hipblasSetMatrixAsync(N, N, sizeof(float), h_a, N, d_a, N, streams[0]);
	hipblasSetMatrixAsync(N, N, sizeof(float), h_b, N, d_b, N, streams[0]);
	hipblasSetMatrixAsync(N, N, sizeof(float), h_c, N, d_c, N, streams[0]);

	hipblasSetMatrixAsync(N, N, sizeof(float), h_x, N, d_x, N, streams[1]);
	hipblasSetMatrixAsync(N, N, sizeof(float), h_y, N, d_y, N, streams[1]);
	hipblasSetMatrixAsync(N, N, sizeof(float), h_z, N, d_z, N, streams[1]);

	// sgemm on streams
    // SGEMM: C = alpha*A*B + beta*C
	hipblasSetStream(handle0, streams[0]);
	hipblasSgemm(handle0, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    N, N, N,
                    &alpha,
                    d_a, N,
                    d_b, N,
                    &beta,
                    d_c, N);

	hipblasSetStream(handle1, streams[1]);
	hipblasSgemm(handle1, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    N, N, N,
                    &alpha,
                    d_x, N,
                    d_y, N,
                    &beta,
                    d_z, N);

	hipblasGetMatrixAsync(N, N, sizeof(float), d_c, N, h_c, N, streams[0]);
	hipblasGetMatrixAsync(N, N, sizeof(float), d_z, N, h_z, N, streams[1]);
	
	hipDeviceSynchronize(); // NOTE: this is needed to make sure prev dev opt is done! 

	// free
	for(int i=0; i<streamsNum; i++) {
		hipStreamDestroy(streams[i]);
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);

	//free(h_a);
	//free(h_b);

	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c);

	hipHostFree(h_x);
	hipHostFree(h_y);
	hipHostFree(h_z);

	return 0;
}
